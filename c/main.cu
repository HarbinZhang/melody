
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <ctime>
#include <chrono>
#include <hipfft/hipfft.h>

#define BUFFER_SIZE 4096

// Complex data type
typedef float2 Complex;

#define SIGNAL_SIZE 1000

typedef struct  WAV_HEADER
{
    /* RIFF Chunk Descriptor */
    uint8_t         RIFF[4];        // RIFF Header Magic header
    uint32_t        ChunkSize;      // RIFF Chunk Size
    uint8_t         WAVE[4];        // WAVE Header
    /* "fmt" sub-chunk */
    uint8_t         fmt[4];         // FMT header
    uint32_t        Subchunk1Size;  // Size of the fmt chunk
    uint16_t        AudioFormat;    // Audio format 1=PCM,6=mulaw,7=alaw,     257=IBM Mu-Law, 258=IBM A-Law, 259=ADPCM
    uint16_t        NumOfChan;      // Number of channels 1=Mono 2=Sterio
    uint32_t        SamplesPerSec;  // Sampling Frequency in Hz
    uint32_t        bytesPerSec;    // bytes per second
    uint16_t        blockAlign;     // 2=16-bit mono, 4=16-bit stereo
    uint16_t        bitsPerSample;  // Number of bits per sample
    /* "data" sub-chunk */
    uint8_t         Subchunk2ID[4]; // "data"  string
    uint32_t        Subchunk2Size;  // Sampled data length
} wav_hdr;
int getFileSize(FILE* inFile);



int main(int argc, char ** argv) {
    wav_hdr wavHeader;
    int headerSize = sizeof(wav_hdr);

    const char* filePath;
    filePath = argv[1];

    FILE* wavFile = fopen(filePath, "r");
    if (wavFile == nullptr)
    {
        fprintf(stderr, "Unable to open wave file: %s\n", filePath);
        return 1;
    }

    
    //Read the header
    size_t bytesRead = fread(&wavHeader, 1, headerSize, wavFile);
    float data_array[wavHeader.Subchunk2Size];
    if (bytesRead > 0)
    {

        //Read the data
        // uint16_t bytesPerSample = wavHeader.bitsPerSample / 8;      //Number     of bytes per sample
        // uint64_t numSamples = wavHeader.ChunkSize / bytesPerSample; //How many samples are in the wav file?
        float* buffer = new float[BUFFER_SIZE];

        int i = 0;
        while ((bytesRead = fread(buffer, sizeof buffer[0], BUFFER_SIZE / (sizeof buffer[0]), wavFile)) > 0)
        {
            /** DO SOMETHING WITH THE WAVE DATA HERE **/
            memcpy(&data_array[BUFFER_SIZE*i], &buffer[0], bytesRead);
            i++;
        }
        delete [] buffer;
        buffer = nullptr;
        printf("%d\n", i);

    }
    fclose(wavFile);

    // int8_t *ginit_array;
    // cudaMalloc((void **) &ginit_array, wavHeader.Subchunk2Size);
    // cudaMemcpy(ginit_array, data_array, wavHeader.Subchunk2Size, cudaMemcpyHostToDevice);

    // int8_t *gout_array;
    // cudaMalloc((void **) &gout_array, wavHeader.Subchunk2Size/ BUFFER_SIZE);
    // cuda_fft<<<1, ceil(wavHeader.Subchunk2Size/BUFFER_SIZE)>>>(ginit_array, gout_array);
    // printf("%s\n", "done");

    printf("[simpleCUFFT] is starting...\n");
    // Allocate host memory for the signal
    // Complex* h_signal = (Complex*)malloc(sizeof(Complex) * SIGNAL_SIZE);
    // Initalize the memory for the signal

    // R 2 C ?
    // 




    return 0;
}

// find the file size
int getFileSize(FILE* inFile)
{
    int fileSize = 0;
    fseek(inFile, 0, SEEK_END);

    fileSize = ftell(inFile);

    fseek(inFile, 0, SEEK_SET);
    return fileSize;
}
