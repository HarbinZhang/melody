#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <ctime>
#include <chrono>
#include <hipfft/hipfft.h>

#define BUFFER_SIZE 4096

// hipfftComplex data type
// typedef float2 hipfftComplex;

#define SIGNAL_SIZE 4096

typedef struct  WAV_HEADER
{
    /* RIFF Chunk Descriptor */
    uint8_t         RIFF[4];        // RIFF Header Magic header
    uint32_t        ChunkSize;      // RIFF Chunk Size
    uint8_t         WAVE[4];        // WAVE Header
    /* "fmt" sub-chunk */
    uint8_t         fmt[4];         // FMT header
    uint32_t        Subchunk1Size;  // Size of the fmt chunk
    uint16_t        AudioFormat;    // Audio format 1=PCM,6=mulaw,7=alaw,     257=IBM Mu-Law, 258=IBM A-Law, 259=ADPCM
    uint16_t        NumOfChan;      // Number of channels 1=Mono 2=Sterio
    uint32_t        SamplesPerSec;  // Sampling Frequency in Hz
    uint32_t        bytesPerSec;    // bytes per second
    uint16_t        blockAlign;     // 2=16-bit mono, 4=16-bit stereo
    uint16_t        bitsPerSample;  // Number of bits per sample
    /* "data" sub-chunk */
    uint8_t         Subchunk2ID[4]; // "data"  string
    uint32_t        Subchunk2Size;  // Sampled data length
} wav_hdr;
int getFileSize(FILE* inFile);

__global__ void all_in(hipfftComplex* in, hipfftComplex* out, int rate);
__global__ void init(hipfftComplex *g);

int main(int argc, char ** argv) {
    wav_hdr wavHeader;
    int headerSize = sizeof(wav_hdr);

    const char* filePath;
    filePath = argv[1];

    FILE* wavFile = fopen(filePath, "r");
    if (wavFile == nullptr)
    {
        fprintf(stderr, "Unable to open wave file: %s\n", filePath);
        return 1;
    }

    
    //Read the header
    size_t bytesRead = fread(&wavHeader, 1, headerSize, wavFile);
    short data_array[wavHeader.Subchunk2Size/2];

    
    if (bytesRead > 0)
    {

        // Read the data
        // uint16_t bytesPerSample = wavHeader.bitsPerSample / 8;      //Number     of bytes per sample
        // uint64_t numSamples = wavHeader.ChunkSize / bytesPerSample; //How many samples are in the wav file?
        int8_t* buffer = new int8_t[BUFFER_SIZE];

        int i = 0;
        while ((bytesRead = fread(buffer, sizeof buffer[0], BUFFER_SIZE / (sizeof buffer[0]), wavFile)) > 0)
        {
            /** DO SOMETHING WITH THE WAVE DATA HERE **/
            memcpy(&data_array[BUFFER_SIZE*i/2], &buffer[0], bytesRead);
            i++;
        }
        delete [] buffer;
        buffer = nullptr;
        printf("%d\n", i);
    }
    fclose(wavFile);


    

    printf("[simpleCUFFT] is starting...\n");
    // Allocate host memory for the signal
    hipfftComplex* h_signal = (hipfftComplex*) malloc(sizeof(hipfftComplex) * wavHeader.Subchunk2Size/2);

    // memcpy(h_signal, &data_array[0], SIGNAL_SIZE);
    for(int i = 0; i < wavHeader.Subchunk2Size/2; i++){
        h_signal[i].x = (float) data_array[i];
        // h_signal[i].x = 1.0f;
        h_signal[i].y = 0.0f;
    }


    auto start = std::chrono::system_clock::now();
    // Initalize the memory for the signal
    int mem_size = sizeof(hipfftComplex) * wavHeader.Subchunk2Size/2;

    // Allocate device memory for signal
    hipfftComplex* g_signal;
    hipMalloc((void**)&g_signal, mem_size);

    // Copy host memory to device
    hipMemcpy(g_signal, h_signal, mem_size,
               hipMemcpyHostToDevice);

    hipfftComplex* g_fft_out;
    hipMalloc((void**)&g_fft_out, sizeof(hipfftComplex) * wavHeader.Subchunk2Size/2);

    // CUFFT plan
    hipfftHandle plan;
    int n[1] = {SIGNAL_SIZE};
    hipfftResult res = hipfftPlanMany(&plan, 1, n,
        NULL, 1, SIGNAL_SIZE,  //advanced data layout, NULL shuts it off
        NULL, 1, SIGNAL_SIZE,  //advanced data layout, NULL shuts it off
        HIPFFT_C2C, wavHeader.Subchunk2Size/2/SIGNAL_SIZE-1);    
        // HIPFFT_C2C, 3);


    // Transform signal and kernel
    // printf("Transforming signal hipfftExecC2C\n");
    hipfftResult err = hipfftExecC2C(plan, (hipfftComplex *)g_signal, (hipfftComplex *)g_fft_out, HIPFFT_FORWARD);    


    // find max fft in fft results.
    hipfftComplex* g_fft_max_out;
    hipMalloc((void**)&g_fft_max_out, sizeof(hipfftComplex) * (wavHeader.Subchunk2Size/2/SIGNAL_SIZE + 1));

    int blockSize = wavHeader.Subchunk2Size/SIGNAL_SIZE/2048 + 1;
    all_in<<<blockSize, wavHeader.Subchunk2Size/2/SIGNAL_SIZE-1>>>(g_fft_out, g_fft_max_out, wavHeader.SamplesPerSec);
    

    // cuda mem copy to host
    hipfftComplex* h_fft;
    h_fft = (hipfftComplex*) malloc(sizeof(hipfftComplex) * (wavHeader.Subchunk2Size/2/SIGNAL_SIZE + 1));

    hipMemcpy(h_fft, g_fft_max_out, sizeof(hipfftComplex) * (wavHeader.Subchunk2Size/2/SIGNAL_SIZE + 1), 
        hipMemcpyDeviceToHost);

    printf("The size is %d\n", (wavHeader.Subchunk2Size/2/SIGNAL_SIZE + 1));


    for(int i = 0; i <= wavHeader.Subchunk2Size/2/SIGNAL_SIZE; i++){
        printf("fft[%d]: %f\n", i, h_fft[i].x);
    }


    init<<<blockSize, wavHeader.Subchunk2Size/2/SIGNAL_SIZE-1>>>(g_fft_max_out);

    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_seconds = end-start;
    printf("Time using in CPU is : %f\n", elapsed_seconds);


    free(h_signal);
    free(h_fft);


    hipfftDestroy(plan);
    hipFree(g_signal);
    hipFree(g_fft_out);
    hipFree(g_fft_max_out);

    return 0;
}


// find the file size
int getFileSize(FILE* inFile)
{
    int fileSize = 0;
    fseek(inFile, 0, SEEK_END);

    fileSize = ftell(inFile);

    fseek(inFile, 0, SEEK_SET);
    return fileSize;
}


__global__ void all_in(hipfftComplex* in, hipfftComplex* out, int rate){
    int index = threadIdx.x + blockIdx.x * 1024;

    // copy to local memory

    // here: optimized
    hipfftComplex local_in[SIGNAL_SIZE];

    for(int i = 0; i < SIGNAL_SIZE; i++){
        local_in[i] = in[i+index*SIGNAL_SIZE];
    }

    // get biggest FFT
    int k = 0;
    float max_fft_value = (local_in[k].x > 0) ? local_in[k].x:-local_in[k].x;
    // printf("%f\n", max_fft_value);
    for(int i = 0; i < SIGNAL_SIZE / 2; i++){
        float curt = (local_in[i].x > 0) ? local_in[i].x:-local_in[i].x;
        if(curt > max_fft_value){
            // printf("%f\n", curt);
            k = i;
            max_fft_value = curt;
        }
    }

    float freq = (k+1) * (float)rate/(float)SIGNAL_SIZE;

    // if(k == 939){
    //     printf("Here!! %f ::: %f \n", max_fft_value, local_in[85].x);
    // }

    out[index].x = (float) freq;
    out[index].y = 0.0f;

}

__global__ void init(hipfftComplex *g)
{
    int i = blockIdx.x;
    int j = threadIdx.x;

    int m = i + blockIdx.z * SIGNAL_SIZE;
    int n = j + blockIdx.y * SIGNAL_SIZE;

    g[j] = sinf(m*m + n);
    __syncthreads();
}