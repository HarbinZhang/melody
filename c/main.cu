
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <ctime>
#include <chrono>
#include <string.h>

#define BUFFER_SIZE 4096

typedef struct  WAV_HEADER
{
    /* RIFF Chunk Descriptor */
    uint8_t         RIFF[4];        // RIFF Header Magic header
    uint32_t        ChunkSize;      // RIFF Chunk Size
    uint8_t         WAVE[4];        // WAVE Header
    /* "fmt" sub-chunk */
    uint8_t         fmt[4];         // FMT header
    uint32_t        Subchunk1Size;  // Size of the fmt chunk
    uint16_t        AudioFormat;    // Audio format 1=PCM,6=mulaw,7=alaw,     257=IBM Mu-Law, 258=IBM A-Law, 259=ADPCM
    uint16_t        NumOfChan;      // Number of channels 1=Mono 2=Sterio
    uint32_t        SamplesPerSec;  // Sampling Frequency in Hz
    uint32_t        bytesPerSec;    // bytes per second
    uint16_t        blockAlign;     // 2=16-bit mono, 4=16-bit stereo
    uint16_t        bitsPerSample;  // Number of bits per sample
    /* "data" sub-chunk */
    uint8_t         Subchunk2ID[4]; // "data"  string
    uint32_t        Subchunk2Size;  // Sampled data length
} wav_hdr;
int getFileSize(FILE* inFile);

__global__ void cuda_fft(int8_t *in, int8_t *out){
    int i = threadIdx.x;

    printf("%d\n", i);
}

int main(int argc, char ** argv) {
    wav_hdr wavHeader;
    int headerSize = sizeof(wav_hdr), filelength = 0;

    const char* filePath;
    filePath = argv[1];

    FILE* wavFile = fopen(filePath, "r");
    if (wavFile == nullptr)
    {
        fprintf(stderr, "Unable to open wave file: %s\n", filePath);
        return 1;
    }

    int8_t data_array[wavHeader.Subchunk2Size];
    //Read the header
    size_t bytesRead = fread(&wavHeader, 1, headerSize, wavFile);
    if (bytesRead > 0)
    {

        //Read the data
        uint16_t bytesPerSample = wavHeader.bitsPerSample / 8;      //Number     of bytes per sample
        uint64_t numSamples = wavHeader.ChunkSize / bytesPerSample; //How many samples are in the wav file?
        int8_t* buffer = new int8_t[BUFFER_SIZE];

        int i = 0;
        while ((bytesRead = fread(buffer, sizeof buffer[0], BUFFER_SIZE / (sizeof buffer[0]), wavFile)) > 0)
        {
            /** DO SOMETHING WITH THE WAVE DATA HERE **/
            memcpy(&data_array[BUFFER_SIZE*i], &buffer[0], bytesRead);
            i++;
        }
        delete [] buffer;
        buffer = nullptr;
        filelength = getFileSize(wavFile);
        printf("%d\n", i);

    }
    fclose(wavFile);

    int8_t *ginit_array;
    hipMalloc((void **) &ginit_array, wavHeader.Subchunk2Size);
    hipMemcpy(ginit_array, data_array, wavHeader.Subchunk2Size, hipMemcpyHostToDevice);

    int8_t *gout_array;
    hipMalloc((void **) &gout_array, wavHeader.Subchunk2Size/ BUFFER_SIZE);
    cuda_fft<<<1, ceil(wavHeader.Subchunk2Size/BUFFER_SIZE)>>>(ginit_array, gout_array);


    return 0;
}

// find the file size
int getFileSize(FILE* inFile)
{
    int fileSize = 0;
    fseek(inFile, 0, SEEK_END);

    fileSize = ftell(inFile);

    fseek(inFile, 0, SEEK_SET);
    return fileSize;
}
