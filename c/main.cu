
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <ctime>
#include <chrono>
#include <string.h>

typedef struct  WAV_HEADER
{
    /* RIFF Chunk Descriptor */
    uint8_t         RIFF[4];        // RIFF Header Magic header
    uint32_t        ChunkSize;      // RIFF Chunk Size
    uint8_t         WAVE[4];        // WAVE Header
    /* "fmt" sub-chunk */
    uint8_t         fmt[4];         // FMT header
    uint32_t        Subchunk1Size;  // Size of the fmt chunk
    uint16_t        AudioFormat;    // Audio format 1=PCM,6=mulaw,7=alaw,     257=IBM Mu-Law, 258=IBM A-Law, 259=ADPCM
    uint16_t        NumOfChan;      // Number of channels 1=Mono 2=Sterio
    uint32_t        SamplesPerSec;  // Sampling Frequency in Hz
    uint32_t        bytesPerSec;    // bytes per second
    uint16_t        blockAlign;     // 2=16-bit mono, 4=16-bit stereo
    uint16_t        bitsPerSample;  // Number of bits per sample
    /* "data" sub-chunk */
    uint8_t         Subchunk2ID[4]; // "data"  string
    uint32_t        Subchunk2Size;  // Sampled data length
} wav_hdr;
int getFileSize(FILE* inFile);


int main(int argc, char ** argv) {
    wav_hdr wavHeader;
    int headerSize = sizeof(wav_hdr), filelength = 0;

    const char* filePath;

    filePath = argv[1];

    FILE* wavFile = fopen(filePath, "r");
    if (wavFile == nullptr)
    {
        fprintf(stderr, "Unable to open wave file: %s\n", filePath);
        return 1;
    }

    //Read the header
    size_t bytesRead = fread(&wavHeader, 1, headerSize, wavFile);
    if (bytesRead > 0)
    {
        //Read the data
        uint16_t bytesPerSample = wavHeader.bitsPerSample / 8;      //Number     of bytes per sample
        uint64_t numSamples = wavHeader.ChunkSize / bytesPerSample; //How many samples are in the wav file?
        static const uint16_t BUFFER_SIZE = 4096;
        int8_t* buffer = new int8_t[BUFFER_SIZE];

        int i = 0;
        while ((bytesRead = fread(buffer, sizeof buffer[0], BUFFER_SIZE / (sizeof buffer[0]), wavFile)) > 0)
        {
            /** DO SOMETHING WITH THE WAVE DATA HERE **/
            i ++;
        }
        delete [] buffer;
        buffer = nullptr;
        filelength = getFileSize(wavFile);
        printf("%d\n", i);

    }
    fclose(wavFile);

    return 0;
}

// find the file size
int getFileSize(FILE* inFile)
{
    int fileSize = 0;
    fseek(inFile, 0, SEEK_END);

    fileSize = ftell(inFile);

    fseek(inFile, 0, SEEK_SET);
    return fileSize;
}
