
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <ctime>
#include <chrono>
#include <hipfft/hipfft.h>

#define BUFFER_SIZE 4096

// Complex data type
typedef float2 Complex;

#define SIGNAL_SIZE 1200

typedef struct  WAV_HEADER
{
    /* RIFF Chunk Descriptor */
    uint8_t         RIFF[4];        // RIFF Header Magic header
    uint32_t        ChunkSize;      // RIFF Chunk Size
    uint8_t         WAVE[4];        // WAVE Header
    /* "fmt" sub-chunk */
    uint8_t         fmt[4];         // FMT header
    uint32_t        Subchunk1Size;  // Size of the fmt chunk
    uint16_t        AudioFormat;    // Audio format 1=PCM,6=mulaw,7=alaw,     257=IBM Mu-Law, 258=IBM A-Law, 259=ADPCM
    uint16_t        NumOfChan;      // Number of channels 1=Mono 2=Sterio
    uint32_t        SamplesPerSec;  // Sampling Frequency in Hz
    uint32_t        bytesPerSec;    // bytes per second
    uint16_t        blockAlign;     // 2=16-bit mono, 4=16-bit stereo
    uint16_t        bitsPerSample;  // Number of bits per sample
    /* "data" sub-chunk */
    uint8_t         Subchunk2ID[4]; // "data"  string
    uint32_t        Subchunk2Size;  // Sampled data length
} wav_hdr;
int getFileSize(FILE* inFile);



int main(int argc, char ** argv) {
    wav_hdr wavHeader;
    int headerSize = sizeof(wav_hdr);

    const char* filePath;
    filePath = argv[1];

    FILE* wavFile = fopen(filePath, "r");
    if (wavFile == nullptr)
    {
        fprintf(stderr, "Unable to open wave file: %s\n", filePath);
        return 1;
    }

    
    //Read the header
    size_t bytesRead = fread(&wavHeader, 1, headerSize, wavFile);
    short data_array[wavHeader.Subchunk2Size];

    auto start = std::chrono::system_clock::now();
    if (bytesRead > 0)
    {

        //Read the data
        // uint16_t bytesPerSample = wavHeader.bitsPerSample / 8;      //Number     of bytes per sample
        // uint64_t numSamples = wavHeader.ChunkSize / bytesPerSample; //How many samples are in the wav file?
        int8_t* buffer = new int8_t[BUFFER_SIZE];

        int i = 0;
        while ((bytesRead = fread(buffer, sizeof buffer[0], BUFFER_SIZE / (sizeof buffer[0]), wavFile)) > 0)
        {
            /** DO SOMETHING WITH THE WAVE DATA HERE **/
            memcpy(&data_array[BUFFER_SIZE*i/2], &buffer[0], bytesRead);
            i++;
        }
        delete [] buffer;
        buffer = nullptr;
        printf("%d\n", i);

    }
    fclose(wavFile);



    printf("[simpleCUFFT] is starting...\n");
    // Allocate host memory for the signal
    // Complex* h_signal = (Complex*)malloc(sizeof(Complex) * SIGNAL_SIZE);
    float* h_signal = (float*) malloc(sizeof(float) * SIGNAL_SIZE);

    // memcpy(h_signal, &data_array[0], SIGNAL_SIZE);
    for(int i = 0; i < SIGNAL_SIZE; i++){
        h_signal[i] = (float) data_array[i];
    }

    for(int i = 0; i < SIGNAL_SIZE; i++){
        printf("%f\n", h_signal[i]);
    }

    // Initalize the memory for the signal
    int mem_size = sizeof(float) * SIGNAL_SIZE;

    // Allocate device memory for signal
    float* g_signal;
    hipMalloc((void**)&g_signal, mem_size);
    // Copy host memory to device
    hipMemcpy(g_signal, h_signal, mem_size,
               hipMemcpyHostToDevice);

    Complex* g_out;
    hipMalloc((void**)&g_out, sizeof(Complex) * SIGNAL_SIZE);

    Complex* h_fft;
    h_fft = (Complex*) malloc(sizeof(Complex) * SIGNAL_SIZE);

    // CUFFT plan
    hipfftHandle plan;
    hipfftPlan1d(&plan, SIGNAL_SIZE, HIPFFT_R2C, 1);

    // Transform signal and kernel
    printf("Transforming signal cufftExecC2C\n");
    hipfftResult err = hipfftExecR2C(plan, (float *)g_signal, (Complex *)g_out);    


    // cuda mem copy to host
    
    hipMemcpy(h_fft, g_out, sizeof(Complex) * SIGNAL_SIZE, 
        hipMemcpyDeviceToHost);


    float* g_signal_out;
    hipMalloc((void**)&g_signal_out, mem_size);

    // Transform signal back
    printf("Transforming signal back cufftExecC2C\n");
    hipfftExecC2R(plan, (Complex *)g_out, (float *)g_signal_out);


    // float* h_out = h_signal;
    float* h_out = (float*) malloc(sizeof(float) * SIGNAL_SIZE);
    hipMemcpy(h_out, g_signal, mem_size, hipMemcpyDeviceToHost);


    for(int i = 0; i < SIGNAL_SIZE; i++){
        printf("fft[%d]: %f\n", i, h_fft[i].x);
    }


    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_seconds = end-start;
    printf("Time using in CPU is : %f\n", elapsed_seconds);
    printf("Error info: %s\n", err);

    hipfftDestroy(plan);

    free(h_signal);
    free(h_fft);

    hipFree(g_signal);
    hipFree(g_out);
    hipFree(g_signal_out);


    return 0;
}

// find the file size
int getFileSize(FILE* inFile)
{
    int fileSize = 0;
    fseek(inFile, 0, SEEK_END);

    fileSize = ftell(inFile);

    fseek(inFile, 0, SEEK_SET);
    return fileSize;
}


__global__ void complex2real(Complex* in, float* out, int N){
    int i = threadIdx.x;
    out[i] = in[i].x / (float)N;
}