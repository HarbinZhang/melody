//includes,project
#include<hip/hip_runtime.h>
#include<hipfft/hipfft.h>
#include<iostream>
using namespace std;
#define SIGNAL_SIZE 16
typedef unsigned int uint32;
typedef unsigned long int uint64;

int main()
{
        //Allocate host memory for the signal
        hipfftComplex* h_signal=(hipfftComplex *)malloc(sizeof(hipfftComplex) *SIGNAL_SIZE);

        //Initialize the memory for the signal
        for(unsigned int i=0;i<SIGNAL_SIZE;++i)
        {
                h_signal[i].x=1;
                h_signal[i].y=0;
        }
        
        //Allocate device memory for signal
        hipfftComplex *d_signal;
        hipMalloc((void **)&d_signal,sizeof(hipfftComplex)*SIGNAL_SIZE);

        //Copy host memory to device
        hipMemcpy(d_signal,h_signal,sizeof(hipfftComplex)*SIGNAL_SIZE,hipMemcpyHostToDevice);

        //CUFFT plan
        hipfftHandle plan;
        // cufftPlan1d(&plan,SIGNAL_SIZE,CUFFT_C2C,1);

        int n[1] = {3};
        hipfftResult res = hipfftPlanMany(&plan, 1, n,
        NULL, 1, 0,  //advanced data layout, NULL shuts it off
        NULL, 1, 0,  //advanced data layout, NULL shuts it off
        HIPFFT_C2C, 4);    

        
        //Transform signal 
        hipfftExecC2C(plan,(hipfftComplex *)d_signal,(hipfftComplex *)d_signal,HIPFFT_FORWARD);

        //Copy device memory to host
        hipMemcpy(h_signal,d_signal,sizeof(hipfftComplex)*SIGNAL_SIZE,hipMemcpyDeviceToHost);

        for(unsigned int i=0;i<SIGNAL_SIZE;++i)
        {
                // cout<<h_signal[i].x<<endl;
                // cout<<h_signal[i].y<<endl;
                printf("%f\n", h_signal[i].x);
        }
        
        //Destory CUFFT context
        hipfftDestroy(plan);
        
        //cleanup memory
        free(h_signal);
        hipFree(d_signal);
        
        hipDeviceReset();
}